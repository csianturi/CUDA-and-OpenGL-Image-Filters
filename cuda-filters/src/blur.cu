#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../include/stb_image.h"
#include "../include/stb_image_write.h"
#include <hip/hip_runtime.h>
#include <chrono>
#include <cstdio>
#include <algorithm>
using namespace std::chrono;

// ─────────────────────────────────────────────────────────────
// CPU 3×3 Box Blur
// ─────────────────────────────────────────────────────────────
void blur_cpu(const unsigned char* in, unsigned char* out,
              int w, int h, int channels)
{
    for (int y = 0; y < h; ++y) {
        for (int x = 0; x < w; ++x) {
            int sum = 0, count = 0;
            for (int dy = -1; dy <= 1; ++dy)
                for (int dx = -1; dx <= 1; ++dx) {
                    // Compute neighbor coordinates
                    int nx = x + dx;
                    int ny = y + dy;

                    // Clamp to image boundaries to avoid out-of-bounds access
                    if (nx < 0) nx = 0;
                    if (ny < 0) ny = 0;
                    if (nx >= w) nx = w - 1;
                    if (ny >= h) ny = h - 1;
                    sum += in[ny * w + nx];
                    ++count;
                }
            out[y * w + x] = static_cast<unsigned char>(sum / count);
        }
    }
}

// ─────────────────────────────────────────────────────────────
// CUDA 3×3 Box Blur
// ─────────────────────────────────────────────────────────────
__global__ void blur_naive(const unsigned char* in, unsigned char* out,
                           int w, int h)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= w || y >= h) return;

    int sum = 0, count = 0;
    for (int dy = -1; dy <= 1; ++dy)
        for (int dx = -1; dx <= 1; ++dx) {
            int nx = min(max(x + dx, 0), w - 1);
            int ny = min(max(y + dy, 0), h - 1);
            sum += in[ny * w + nx];
            ++count;
        }
    out[y * w + x] = static_cast<unsigned char>(sum / count);
}

// ─────────────────────────────────────────────────────────────
// Main
// ─────────────────────────────────────────────────────────────
int main()
{
    // ─── Load grayscale image ───
    int width, height, channels;
    unsigned char* h_input = stbi_load("../images/input_gray.png", &width, &height, &channels, 1);
    if (!h_input) {
        printf("Failed to load image.\n");
        return -1;
    }
    printf("Loaded image: %dx%d\n", width, height);

    size_t imgSize = width * height;
    unsigned char* h_output_cpu = (unsigned char*)malloc(imgSize);
    unsigned char* h_output_gpu = (unsigned char*)malloc(imgSize);

    // ─── CPU BLUR ───
    auto t1 = high_resolution_clock::now();
    blur_cpu(h_input, h_output_cpu, width, height, 1);
    auto t2 = high_resolution_clock::now();
    double cpu_ms = duration<double, std::milli>(t2 - t1).count();
    printf("CPU blur: %.3f ms\n", cpu_ms);

    stbi_write_png("../images/output_blur_cpu.png", width, height, 1, h_output_cpu, width);
    printf("Wrote CPU blurred image -> output_blur_cpu.png\n");

    // ─── GPU BLUR ───
    unsigned char *d_input, *d_output;
    hipMalloc(&d_input, imgSize);
    hipMalloc(&d_output, imgSize);

    // CUDA event setup for timing
    hipEvent_t eStart, eAfterH2D, eAfterKernel, eStop;
    hipEventCreate(&eStart);
    hipEventCreate(&eAfterH2D);
    hipEventCreate(&eAfterKernel);
    hipEventCreate(&eStop);

    hipEventRecord(eStart);
    hipMemcpy(d_input, h_input, imgSize, hipMemcpyHostToDevice);
    hipEventRecord(eAfterH2D);

    dim3 block(16, 16); // or try (32,8)
    dim3 grid((width + block.x - 1) / block.x,
              (height + block.y - 1) / block.y);

    blur_naive<<<grid, block>>>(d_input, d_output, width, height);
    hipDeviceSynchronize();
    hipEventRecord(eAfterKernel);

    hipMemcpy(h_output_gpu, d_output, imgSize, hipMemcpyDeviceToHost);
    hipEventRecord(eStop);
    hipEventSynchronize(eStop);

    // ─── Measure CUDA times ───
    float tH2D=0, tKernel=0, tD2H=0, tTotal=0;
    hipEventElapsedTime(&tH2D, eStart, eAfterH2D);
    hipEventElapsedTime(&tKernel, eAfterH2D, eAfterKernel);
    hipEventElapsedTime(&tD2H, eAfterKernel, eStop);
    hipEventElapsedTime(&tTotal, eStart, eStop);

    printf("\n--- Timing (ms) ---\n");
    printf("CPU blur     : %.3f\n", cpu_ms);
    printf("H2D transfer : %.3f\n", tH2D);
    printf("Kernel       : %.3f\n", tKernel);
    printf("D2H transfer : %.3f\n", tD2H);
    printf("Total GPU    : %.3f\n", tTotal);
    printf("Speedup (CPU / Total GPU): %.2fx\n", cpu_ms / tTotal);

    // ─── Validate output ───
    int mismatches = 0;
    for (int i = 0; i < width * height; ++i) {
        int diff = int(h_output_cpu[i]) - int(h_output_gpu[i]);
        if (abs(diff) > 1) ++mismatches;
    }
    printf("Mismatches: %d\n", mismatches);

    stbi_write_png("../images/output_blur_gpu.png", width, height, 1, h_output_gpu, width);
    printf("Wrote GPU blurred image -> output_blur_gpu.png\n");

    // ─── Cleanup ───
    hipFree(d_input);
    hipFree(d_output);
    free(h_output_cpu);
    free(h_output_gpu);
    stbi_image_free(h_input);
    hipEventDestroy(eStart);
    hipEventDestroy(eAfterH2D);
    hipEventDestroy(eAfterKernel);
    hipEventDestroy(eStop);

    return 0;
}
